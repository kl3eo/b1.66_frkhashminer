#include "hip/hip_runtime.h"
/* Copyright (C) 1883 Thomas Edison - All Rights Reserved
 * You may use, distribute and modify this code under the
 * terms of the GPLv3 license, which unfortunately won't be
 * written for another century.
 *
 * You should have received a copy of the LICENSE file with
 * this file.
 */

#include "frkhash_cuda_miner_kernel.h"

#include "frkhash_cuda_miner_kernel_globals.h"

#include "cuda_helper.h"

#define copy(dst, src, count)                                                                                          \
    for (int i = 0; i != count; ++i) {                                                                                 \
        (dst)[i] = (src)[i];                                                                                           \
    }

#include "keccak.cuh"

#include "hash_it.cuh"

__global__ void frkhash_search(Search_results* g_output, uint64_t start_nonce) {
    if (g_output->done)
        return;
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    bool r = compute_hash(start_nonce + gid);
    if (threadIdx.x == 0)
        atomicInc((uint32_t*)&g_output->hashCount, 0xffffffff);
    if (r)
        return;
    uint32_t index = atomicInc((uint32_t*)&g_output->solCount, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;
    g_output->gid[index] = gid;
    g_output->done = 1;
}

void run_frkhash_search(uint32_t gridSize, uint32_t blockSize, hipStream_t stream, Search_results* g_output,
                       uint64_t start_nonce) {
    frkhash_search<<<gridSize, blockSize, 0, stream>>>(g_output, start_nonce);
    CUDA_CALL(hipGetLastError());
}

void set_header(hash32_t _header) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t))); }

void set_target(uint64_t _target) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t))); }